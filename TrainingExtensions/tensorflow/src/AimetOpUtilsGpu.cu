
#include <hip/hip_runtime.h>
//==============================================================================
//
//  @@-COPYRIGHT-START-@@
//
//  Copyright (c) 2020-2022, Qualcomm Innovation Center, Inc. All rights reserved.
//
//  Redistribution and use in source and binary forms, with or without
//  modification, are permitted provided that the following conditions are met:
//
//  1. Redistributions of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//  2. Redistributions in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//  3. Neither the name of the copyright holder nor the names of its contributors
//     may be used to endorse or promote products derived from this software
//     without specific prior written permission.
//
//  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
//  AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
//  IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
//  ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
//  LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
//  CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
//  SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
//  INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
//  CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
//  ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
//  POSSIBILITY OF SUCH DAMAGE.
//
//  SPDX-License-Identifier: BSD-3-Clause
//
//  @@-COPYRIGHT-END-@@
//
//==============================================================================

#ifdef GOOGLE_CUDA

#define EIGEN_USE_GPU
#define EIGEN_USE_THREADS

#include "AimetOpUtils.h"

using namespace tensorflow;

#define EIGEN_USE_GPU
typedef Eigen::GpuDevice GPUDevice;


// GPU specialization of actual computations.
template <typename T>
void copyInputTensorsToOutputTensors(const GPUDevice& d, const T* inTensor, size_t count, T* outTensor)
{
    // copy input_tensor to output_tensor
    cudaMemcpy(outTensor, inTensor, count * sizeof(float), cudaMemcpyDeviceToDevice);
}

template <typename T>
T copyLiteralToHost(const GPUDevice& d, const T* deviceValue)
{
    T hostValue;
    cudaMemcpy(&hostValue, deviceValue, sizeof(T), cudaMemcpyDeviceToHost);

    return hostValue;
}

void sliceTensorAlongLastDim(const GPUDevice& d, Tensor slicedTensor, const Tensor& tensorToSlice, int channel)
{
    // K x K x I x O -> N x O
    auto tensorToSliceTwoDim = tensorToSlice.flat_inner_dims<float, 2>();
    slicedTensor.tensor<float, 2>().chip<0>(0).device(d) = tensorToSliceTwoDim.chip<1>(channel);

}

void sliceAndStoreTensor(const GPUDevice& d, Tensor* slicedTensor, Tensor tensorToSlice, int channel)
{
    auto slicedTensorTwoDim = slicedTensor->flat_inner_dims<float, 2>();
    slicedTensorTwoDim.chip<1>(channel).device(d) = tensorToSlice.tensor<float, 2>().chip<0>(0);
}

template void copyInputTensorsToOutputTensors(const GPUDevice& d, const float* inTensor, size_t count, float* outTensor);
template int8 copyLiteralToHost<int8>(const GPUDevice&, const int8* deviceValue);
template int32 copyLiteralToHost<int32>(const GPUDevice&, const int32* deviceValue);
template uint64 copyLiteralToHost<uint64>(const GPUDevice&, const uint64* deviceValue);
template double copyLiteralToHost<double>(const GPUDevice&, const double* deviceValue);
template bool copyLiteralToHost<bool>(const GPUDevice&, const bool* deviceValue);

#endif   // GOOGLE_CUDA